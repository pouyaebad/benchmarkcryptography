//
// Copyright 2024, Pouya Ebadollahyvahed
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files(the �Software�),
// to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, 
// sublicense, and /or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following 
// conditions :
//
// The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF 
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY 
// CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
// SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//

#include <cassert>

#include "utility.h"
#include "cuda_main.cuh"

/**********************************************************************************************************************************/
/**********************************************************************************************************************************/
/*                                                                                                                                */
/*                                                      CUDA_Features                                                             */
/*                                                                                                                                */
/*                                                   Class Member Functions                                                       */
/*                                                                                                                                */
/* A helper class to get GPU�s hardware information                                                                               */
/*                                                                                                                                */
/**********************************************************************************************************************************/


CUDA_Features::CUDA_Features() : m_device_count{ }
{
    if (hipSuccess != hipGetDeviceCount(&m_device_count))
        m_device_count = 0;
}


int CUDA_Features::CUDA_Get_Total_GPUs_Count() const
{
    return m_device_count;
}


std::vector<std::string> CUDA_Features::CUDA_Get_GPU_Information(const int deviceNo) const
{
    const int info_field_count = 12;

    std::vector<std::string> ret;


    if (deviceNo >= m_device_count)  // updating the member variable
        return ret;


    hipDeviceProp_t cudaProp;
    if (hipSuccess != hipGetDeviceProperties(&cudaProp, deviceNo))
    {
        ret.push_back(" CUDA Device Query Failed");               // Alternative #1

        for (int i = 0; i < info_field_count - 1; i++)
            ret.push_back("-");

        return ret;
    }


    std::string sData{ cudaProp.name };
    if (m_device_count > 1)
        sData += "  ( + " + std::to_string(m_device_count - 1) + " More Devices )";

    ret.push_back(sData);                                                                                // Item #1
    ret.push_back(format_number_3digits_n_suffix(cudaProp.totalGlobalMem, 1024LL) + "B");                // Item #2


    sData = std::to_string(cudaProp.multiProcessorCount);
    int iCores = CUDA_Get_CUDA_Cores_Count(cudaProp.major, cudaProp.minor);

    if (iCores > 0)
        sData += "  ( " + std::to_string(iCores * cudaProp.multiProcessorCount) + " Cores or Thread/Stream Processor)";

    ret.push_back(sData);                                                                               // Item #3


    ret.push_back(format_number_3digits_n_suffix(1000LL * (size_t)cudaProp.clockRate) + "Hz");          // Item #4
    ret.push_back(format_number_3digits_n_suffix(1000LL * (size_t)cudaProp.memoryClockRate) + "Hz");    // Item #5
    ret.push_back(std::to_string(cudaProp.regsPerMultiprocessor));                                      // Item #6
    ret.push_back(std::to_string(cudaProp.regsPerBlock));                                               // Item #7
    ret.push_back(std::to_string(cudaProp.maxGridSize[0]) + ", " + std::to_string(cudaProp.maxGridSize[1]) + ", " + std::to_string(cudaProp.maxGridSize[2]));               // Item #8 
    ret.push_back(std::to_string(cudaProp.maxBlocksPerMultiProcessor));                                 // Item #9
    ret.push_back(std::to_string(cudaProp.maxThreadsPerBlock));                                         // Item #10
    ret.push_back(std::to_string(cudaProp.maxThreadsDim[0]) + ", " + std::to_string(cudaProp.maxThreadsDim[1]) + ", " + std::to_string(cudaProp.maxThreadsDim[2]));         // Item #11
    ret.push_back(std::to_string(cudaProp.maxThreadsPerMultiProcessor));                                // Item #12

    assert(ret.size() == info_field_count);

    // hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
    if (hipSuccess != hipDeviceReset())
        ret.clear();

    return ret;
}
