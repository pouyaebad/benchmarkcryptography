#include "hip/hip_runtime.h"
//
// Copyright 2024, Pouya Ebadollahyvahed
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files(the �Software�),
// to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, 
// sublicense, and /or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following 
// conditions :
//
// The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF 
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY 
// CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
// SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//

#include "AES_GCM_GPU.cuh"
#include "AES_GCM_IMPL.cuh"


/**********************************************************************************************************************************/
/**********************************************************************************************************************************/
/*                                                                                                                                */
/*                                                    CUDA Kernel Function                                                        */
/*                                                                                                                                */
/*                                              Called from Host but Runs on Device                                               */
/*                                                        Multi-Threaded                                                          */
/*                                                                                                                                */
/* CUDA kernel creates copies of AES_GCM_IMPL so it can run AES-GCM on GPU                                                        */
/*                                                                                                                                */
/**********************************************************************************************************************************/

__global__ void run_Encyption_Decryption_on_Vector_GPU_Kernel(uint08T* bufDataVectorMaster, uint08T* bufDataOutput, size_t* returnedStatParameters)
{
	bool log_enabled{ false };
	long execution_rounds{ (long)returnedStatParameters[7] };

	uint08T*		bufDataVector, recordType, ct_buf[256], pt_buf[256], tag_buf[256]; // 16 bytes is enough for TAG but I considered 256 bytes for robustness
	GCM_CNTX		ctx;
	AES_GCM_IMPL	aes_gcm_impl;

	AES_GCM_PARAMS<uint08T>	aes_gcm_params;

	size_t	v_encryption_operations_count{}, v_dencryption_operations_successful_count{}, v_authentication_failed_operations_count{},
		v_128bit_operations_count{}, v_192bit_operations_count{}, v_256bit_operations_count{}, v_aes_processed_messages_total_length{};

	size_t	gcm_context_size{ sizeof(GCM_CNTX) };

	if ((0 == blockIdx.x) && (0 == threadIdx.x) && (NULL != bufDataOutput) && (NULL != returnedStatParameters))
			log_enabled = true;


	for (long execution_index = 0L; execution_index < execution_rounds; execution_index++)
	{
		bufDataVector = bufDataVectorMaster;

		// Running for the whole Test-Data_Buffer (vector) Once
		// vector has multiple recrds, each records starts with "Record Type" Byte and then the record itself
		while (true)
		{
			bufDataVector = getOneRecordofData(bufDataVector, aes_gcm_params, recordType);

			if (0 == recordType) // Record Type 0 means end of file
			{
				if (true == log_enabled)
					if (0L == execution_index)
						*bufDataOutput = 0;
				break;
			}


			aes_gcm_impl.gcm_setkey(&ctx, aes_gcm_params.key, (const uint32T)aes_gcm_params.keyLength);// returns false if key length is not 128, 192 or 256 bit


			switch (recordType)   // based upon our record type, run a test...
			{
			case 1:     // Encryption
				aes_gcm_impl.gcm_start(&ctx, MODE_ENCRYPT, aes_gcm_params.initVector, aes_gcm_params.initVectorLength, aes_gcm_params.assocAuth, aes_gcm_params.assocAuthLength);
				aes_gcm_impl.gcm_process(&ctx, aes_gcm_params.plainTextLength, aes_gcm_params.plainText, ct_buf);
				aes_gcm_impl.gcm_finish(&ctx, tag_buf, aes_gcm_params.authTagLength);
				memset(&ctx, 0, gcm_context_size);


				if (true == log_enabled)
					if (0L == execution_index)
					{
						v_encryption_operations_count++;
						v_aes_processed_messages_total_length += (size_t)aes_gcm_params.plainTextLength;

						if (16 == (size_t)aes_gcm_params.keyLength)
							v_128bit_operations_count++;
						else if (24 == (size_t)aes_gcm_params.keyLength)
							v_192bit_operations_count++;
						else if (32 == (size_t)aes_gcm_params.keyLength)
							v_256bit_operations_count++;

						// writing output buffer but replacing ciphered text with our own calculated one
						bufDataOutput = setOneRecordofData(bufDataOutput, aes_gcm_params, recordType, NULL, ct_buf);
					}
				break;

			case 2:   // Decryption OK
			case 3:   // Decryption Failed Because of Authetication (as per Plan)
				aes_gcm_impl.gcm_start(&ctx, MODE_DECRYPT, aes_gcm_params.initVector, aes_gcm_params.initVectorLength, aes_gcm_params.assocAuth, aes_gcm_params.assocAuthLength);
				aes_gcm_impl.gcm_process(&ctx, aes_gcm_params.CipherTextLength, aes_gcm_params.CipherText, pt_buf);
				aes_gcm_impl.gcm_finish(&ctx, tag_buf, aes_gcm_params.authTagLength);
				memset(&ctx, 0, gcm_context_size);


				if (true == log_enabled)
					if (0L == execution_index)
					{
						v_aes_processed_messages_total_length += (size_t)aes_gcm_params.CipherTextLength;

						if (2 == recordType)  // Decryption OK
							v_dencryption_operations_successful_count++;
						else
							v_authentication_failed_operations_count++;


						if (16 == (size_t)aes_gcm_params.keyLength)
							v_128bit_operations_count++;
						else if (24 == (size_t)aes_gcm_params.keyLength)
							v_192bit_operations_count++;
						else if (32 == (size_t)aes_gcm_params.keyLength)
							v_256bit_operations_count++;

						// writing output buffer but replacing plain text with our own calculated one
						bufDataOutput = setOneRecordofData(bufDataOutput, aes_gcm_params, recordType, pt_buf, NULL);


						// Checking authentication result
						/*
						int index{}, diff{};
						for (; index < aes_gcm_params.authTagLength; index++) // now we verify the authentication tag with generated tag
							diff |= (tag_buf[index] ^ aes_gcm_params.authTag[index]);


						if (((0 != diff) && (2 == recordType)) || ((0 == diff) && (3 == recordType)))
						{
							returnedStatParameters[8] = 0x7FFF;
							return;
						}
						*/
					}
				break;

			default: // we should not have any other record type
				returnedStatParameters[8] = 0x7FFF;
				return;

				break;
			}
		}
	}


	if (true == log_enabled)
	{
		returnedStatParameters[0] = v_encryption_operations_count;
		returnedStatParameters[1] = v_dencryption_operations_successful_count;
		returnedStatParameters[2] = v_authentication_failed_operations_count;
		returnedStatParameters[3] = v_128bit_operations_count;
		returnedStatParameters[4] = v_192bit_operations_count;
		returnedStatParameters[5] = v_256bit_operations_count;
		returnedStatParameters[6] = v_aes_processed_messages_total_length;
	}
}


/**********************************************************************************************************************************/
/**********************************************************************************************************************************/
/*                                                                                                                                */
/*                                                       AES_GCM_GPU                                                              */
/*                                                                                                                                */
/*                                                   Class Member Functions                                                       */
/*                                                                                                                                */
/* Inherited from AES_GCM, runs CUDA kernel. CUDA kernel creates copies of AES_GCM_IMPL so it can run AES-GCM on GPU              */
/*                                                                                                                                */
/**********************************************************************************************************************************/


AES_GCM_GPU::AES_GCM_GPU() : AES_GCM()  // Default Constructor
{
}


AES_GCM_GPU::AES_GCM_GPU(const AES_GCM_GPU& aes_gcm) : AES_GCM(aes_gcm) // Copy Constructor
{
}


AES_GCM_GPU& AES_GCM_GPU::operator=(const AES_GCM_GPU& rhs)  // Assignment operator
{
	if (this != &rhs)
		AES_GCM::operator=(rhs);

	return *this;
}


AES_GCM_GPU::AES_GCM_GPU(AES_GCM_GPU&& aes_gcm) noexcept : AES_GCM(std::move(aes_gcm))  // Move Constructor
{
}


AES_GCM_GPU& AES_GCM_GPU::operator=(AES_GCM_GPU&& rhs) noexcept  // Move Assignment operator
{
	if (this != &rhs)
		AES_GCM::operator=(std::move(rhs));

	return *this;
}



std::string AES_GCM_GPU::get_platform_name() const
{
	int device_count{};
	std::string res{ "GPU (not Detected)" };

	if (hipSuccess != hipGetDeviceCount(&device_count))
		return res;


	hipDeviceProp_t cudaProp;
	if (hipSuccess != hipGetDeviceProperties(&cudaProp, m_gpu_number_to_run))
		return res;

	res = "GPU, ";
	res += cudaProp.name;

	return res;
}



std::pair<long, long> AES_GCM_GPU::get_processing_cores_total() const
{
	int device_count{};

	if (hipSuccess != hipGetDeviceCount(&device_count))
		return std::make_pair(0L, 0L);

	hipDeviceProp_t cudaProp;
	if (hipSuccess != hipGetDeviceProperties(&cudaProp, m_gpu_number_to_run))
		return std::make_pair(0L, 0L);


	int iCores = CUDA_Get_CUDA_Cores_Count(cudaProp.major, cudaProp.minor);

	if (iCores <= 0)
		iCores = 128; // for any reason if CUDA cannot determine correct cores per SM, I assume 128 cores per SM which is very common one


	hipDeviceReset(); // hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.

	return std::make_pair((long)(cudaProp.multiProcessorCount), (long)iCores);
}



int AES_GCM_GPU::get_active_gpu_number() const
{
	return m_gpu_number_to_run;
}



bool AES_GCM_GPU::set_active_gpu_number(const int gpu_number)
{
	if (hipSuccess != hipSetDevice(gpu_number))
		return false;

	m_gpu_number_to_run = gpu_number;

	hipDeviceReset();

	return true;
}



void AES_GCM_GPU::run_Benchmark_core(const long threads_blocks_count, const long threads_per_block_count, const long execution_rounds)
{
	hipError_t	cudaStatus;
	uint08T*	bufDataVector, * bufDataOutput, * dev_bufDataVector, * dev_bufDataOutput;
	size_t		returnedStatParameters[9]{}, * dev_returnedStatParameters; // to see definition of each element of this array, go to end of this function

	returnedStatParameters[7] = execution_rounds;

	bufDataVector = m_buffer_AES_GCM_Vector.get();
	bufDataOutput = m_buffer_AES_GCM_Output.get();


	// Since CUDA 12, calling hipSetDevice() is enough for the whole CUDA initilization
	if (hipSuccess != (cudaStatus = hipSetDevice(m_gpu_number_to_run)))
	{
		std::exception e("Error #AGG-01: CUDA Compatible GPU / Device is not Found");
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipMalloc((void**)&dev_bufDataVector, m_buffer_Size * sizeof(uint08T))))
	{
		std::exception e("Error #AGG-02: CUDA Memory Allocation is Failed for Vector Data");
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(dev_bufDataVector, bufDataVector, m_buffer_Size * sizeof(uint08T), hipMemcpyHostToDevice)))
	{
		hipFree(dev_bufDataVector);

		std::exception e("Error #AGG-03: Copy From System Memory to CUDA Memory is Failed");
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipMalloc((void**)&dev_bufDataOutput, m_buffer_Size * sizeof(uint08T))))
	{
		hipFree(dev_bufDataVector);

		std::exception e("Error #AGG-04: CUDA Memory Allocation is Failed for Output Result Data");
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipMalloc((void**)&dev_returnedStatParameters,  sizeof(returnedStatParameters))))
	{
		hipFree(dev_bufDataVector);
		hipFree(dev_bufDataOutput);

		std::exception e("Error #AGG-05: CUDA Memory Allocation is Failed for Result Data");
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(dev_returnedStatParameters, returnedStatParameters, sizeof(returnedStatParameters), hipMemcpyHostToDevice)))
	{
		hipFree(dev_bufDataVector);
		hipFree(dev_bufDataOutput);
		hipFree(dev_returnedStatParameters);

		std::exception e("Error #AGG-06: Copy From System Memory to CUDA Memory is Failed");
		throw e;
	}



	// ============================================================
	// Creating CUDA Threads & Running AES-GCM operations on them
	// ============================================================

	run_Encyption_Decryption_on_Vector_GPU_Kernel << < threads_blocks_count, threads_per_block_count >> > (dev_bufDataVector, dev_bufDataOutput, dev_returnedStatParameters);

	if (hipSuccess != (cudaStatus = hipGetLastError()))  // Check for any errors launching the kernel
	{
		hipFree(dev_bufDataVector);
		hipFree(dev_bufDataOutput);
		hipFree(dev_returnedStatParameters);

		std::string s = "Error #AGG-07: CUDA Threads Running Failed: ";
		s += hipGetErrorString(cudaStatus);
		std::exception e(s.data());
		throw e;
	}

	if (hipSuccess != (cudaStatus = hipDeviceSynchronize())) // waiting for the kernel (threads) to finish
	{
		hipFree(dev_bufDataVector);
		hipFree(dev_bufDataOutput);
		hipFree(dev_returnedStatParameters);

		std::string s = "Error #AGG-08: CUDA Threads Running Failed: ";
		s += hipGetErrorString(cudaStatus);
		std::exception e(s.data());
		throw e;
	}

	hipFree(dev_bufDataVector);


	if (hipSuccess != (cudaStatus = hipMemcpy(bufDataOutput, dev_bufDataOutput, m_buffer_Size * sizeof(uint08T), hipMemcpyDeviceToHost)))
	{
		hipFree(dev_bufDataOutput);
		hipFree(dev_returnedStatParameters);

		std::exception e("Error #AGG-09: Copy From CUDA Memory to System Memory is Failed");
		throw e;
	}

	hipFree(dev_bufDataOutput);


	if (hipSuccess != (cudaStatus = hipMemcpy(returnedStatParameters, dev_returnedStatParameters, sizeof(returnedStatParameters), hipMemcpyDeviceToHost)))
	{
		hipFree(dev_returnedStatParameters);

		std::exception e("Error #AGG-10: Copy From CUDA Memory to System Memory is Failed");
		throw e;
	}

	hipFree(dev_returnedStatParameters);


	m_encryption_operations_count = returnedStatParameters[0];
	m_dencryption_operations_successful_count = returnedStatParameters[1];
	m_authentication_failed_operations_count = returnedStatParameters[2];
	m_128bit_operations_count = returnedStatParameters[3];
	m_192bit_operations_count = returnedStatParameters[4];
	m_256bit_operations_count = returnedStatParameters[5];
	m_aes_processed_messages_total_length = returnedStatParameters[6];
	//returnedStatParameters[7], it is input to kernel function. it is execution_rounds
	//returnedStatParameters[8], it is set to 0 here and any kernel thread can set it to nonzero if it cannot run properly


	// hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
	if (hipSuccess != (cudaStatus = hipDeviceReset()))
	{
		std::exception e("Error #AGG-11: Resetting CUDA Device is Failed");
		throw e;
	}

	if (0 != returnedStatParameters[8])
	{
		std::exception e("Error #AGG-12: Execution of Threads for AES-GCM GPU Operation Encountered some Errors");
		throw e;
	}
}
